#include "hip/hip_runtime.h"
/* Pi - CUDA version 1 - uses integers for CUDA kernels
 * Author: Felipe Gutierrez, SBEL, July 2015
 */
#include <iostream>
#include <stdio.h> /* fprintf()  */
#include <cstdlib> /*  malloc and free */
#include <float.h> /* DBL_EPSILON() */
#include <math.h> /* sqrt() */
#include <ctime>

#include "pi-kernel.h"

/* Only add openmp if it will be used */
#if OPENMP_ENABLED
    #include <omp.h>
#endif


/**
 * @brief CUDA macro
 * @details 
 *         If CUDA is enabled we need to define:
 *         * nthreads = Number of threads per block we want.
 *         
 *         * NUMBLOCKS = Gives the number of blocks we want to use to parallelize a problem of 
 *         size n.
 *         
 *         * KERNEL =  KERNEL(n) to specified the number of blocks and the number of threads 
 *         per block if CUDA is ENABLED. If CUDA is not enabled then KERNEL(n) is just an empty 
 *         piece of code.
 * 
 */
#if CUDA_ENABLED
    #include "TimerGPU.h"
    #include <thrust/reduce.h>
    #include <thrust/system/cuda/execution_policy.h>
    #include <thrust/system/omp/execution_policy.h>
    #define nthreads 1024
    #define getGridDim(n) (int)ceil(sqrt(n/nthreads))
    #define GRID(n) dim3(getGridDim(n), getGridDim(n))
    #define BLOCK(n) dim3(nthreads)
    #define KERNEL(n) <<<GRID(n), BLOCK(n)>>> /* Necessary for kernels */
#else
    #include "TimerCPU.h"
    #define KERNEL(n) /* Empty code */
#endif

/**
 * @brief calculateAreas kernel
 * @details 
 *         * threadId: Index in the areas area. Tells us where to store the calculated area. With 
 *         CUDA this is calculated with threadId and blockId. In serial and OpenMP this is the 
 *         obtained by the for loop counter.       
 *         * x: Current x coordinate
 *         * heightSq: height of rectangle squared
 * 
 * @param numRects numRects we are going to use to estimate the area under the curve. This defines
 * how big our problem size will be. This is the n in KERNEL(n).
 * 
 * @param width of rectangle
 * 
 * @param areas Pre allocated array that will contain  areas. --> This array was allocated with 
 * hipMallocManaged() function which is what leads to UnifiedMemory.
 * 
 * @return fills the areas array
 */

#if CUDA_ENABLED
__global__ 
#endif
void calculateAreas(const long numRects, const double width, double *dev_areas) 
{
/* If cuda is enabled calculate the threadId which gives us the index in dev_areas */   
#if CUDA_ENABLED
    /* Calculate threadId for 1D grid 1D block*/
    //int threadId = threadIdx.x + (blockIdx.x * blockDim.x);
    /* Calculate threadId for 2D grid 1D block*/
    int threadId = (blockIdx.y*gridDim.x + blockIdx.x)*blockDim.x + threadIdx.x;
    if(threadId >= numRects)
    {
        return;
    }
#else
    /* We don't have to delete the #pragma clause for the serial version of the code. If the program is not compiled with -fopenmp and omp.h is not included the compiler will just ignore the #pragma clause. */
    #pragma omp parallel for
    /* Define the for loop if cuda is not enable. This is used in both the serial and openmp version */
    for(int threadId = 0;threadId < numRects;threadId++)
#endif
    {
        double x = threadId * width;
        double heightSq = 1 - (x*x);
        double height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt(heightSq));
        dev_areas[threadId] = (width * height); 
    }
}

void calculateArea(const long numRects, double *area) 
{
    double *unifiedAreas;
    int i;

/////////////////////////////// MEMORY ALLOCATION SECTION ////////////////////////////////////////

/* If CUDA is enabled allocate memory in device either using hipMalloc or hipMallocManaged */
    hipError_t err;

    if(getGridDim(numRects) >= 65535)
    {
        fprintf(stderr, "Error: WAY TOO MANY RECTANGLES. Do you really want to compute more than 4.3979123e+12 rectangles!!!! Please input less rectangles");
        return;
    }

    std::cout << "Grid Dimensions = " << getGridDim(numRects) << std::endl;
    printf("Unified Memory is Enabled. Allocating using hipMallocManaged \n");
    err = hipMallocManaged(&unifiedAreas, numRects * sizeof(double));

    /* Check for error in device memory allocation */
    if (err != hipSuccess) 
    {
        fprintf(stderr, "hipMalloc or hipMallocManaged failed: %s\n", hipGetErrorString(err));
    }
    
/////////////////////////////// KERNEL CALL SECTION ////////////////////////////////////////

/* If CUDA is enabled do the kernel and reduce call either with unifiedMemory or with device memory*/
/* If CUDA is not enabled calculateAreas is not a kernel but a normal function. */

    calculateAreas KERNEL(numRects) (numRects, (1.0 / numRects), unifiedAreas);

    (*area) = 0.0;

#if CUDA_ENABLED
    (*area) = thrust::reduce(thrust::cuda::par, unifiedAreas, unifiedAreas + numRects);
#else
    for (i = 0; i < numRects; i++) 
    {
        (*area) += unifiedAreas[i];
    }
#endif

    hipFree(unifiedAreas);

///////////////////// GPU OR CPU FREE THE MEMORY ////////////////////

}

#if CUDA_ENABLED
void printDeviceInfo()
{
    int device;
    struct hipDeviceProp_t props;

    hipGetDevice(&device);
    hipGetDeviceProperties(&props, device);

    std::cout << "Device info: " <<std::endl;
    std::cout << "Name: " << props.name <<std::endl;
    std::cout << "version: " << props.major << "," <<  props.minor <<std::endl;
}
#endif