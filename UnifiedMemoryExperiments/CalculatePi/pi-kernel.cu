#include "hip/hip_runtime.h"
/* Pi - CUDA version 1 - uses integers for CUDA kernels
 * Author: Felipe Gutierrez, Shodor, May 2015
 */
#include <iostream>
#include <stdio.h> /* fprintf()  */
#include <cstdlib> /*  malloc and free */
#include <float.h> /* DBL_EPSILON() */
#include <math.h> /* sqrt() */


#include "pi-kernel.h"

/* Only add openmp if it will be used */
#if OPENMP_ENABLED
    #include <omp.h>
#endif


/**
 * @brief CUDA macro
 * @details 
 *         If CUDA is enabled we need to define:
 *         * nthreads = Number of threads per block we want.
 *         
 *         * NUMBLOCKS = Gives the number of blocks we want to use to parallelize a problem of 
 *         size n.
 *         
 *         * KERNEL =  KERNEL(n) to specified the number of blocks and the number of threads 
 *         per block if CUDA is ENABLED. If CUDA is not enabled then KERNEL(n) is just an empty 
 *         piece of code.
 * 
 */
#if CUDA_ENABLED
    #include <thrust/reduce.h>
    #include <thrust/system/cuda/execution_policy.h>
    #include <thrust/system/omp/execution_policy.h>
    #define nthreads 1024
    #define getGridDim(n) (int)ceil(sqrt(n/nthreads))
    #define GRID(n) dim3(getGridDim(n), getGridDim(n))
    #define BLOCK(n) dim3(nthreads)
    #define KERNEL(n) <<<GRID(n), BLOCK(n)>>> /* Necessary for kernels */
#else
    #define KERNEL(n) /* Empty code */
#endif

/**
 * @brief calculateAreas kernel
 * @details 
 *         * threadId: Index in the areas area. Tells us where to store the calculated area. With 
 *         CUDA this is calculated with threadId and blockId. In serial and OpenMP this is the 
 *         obtained by the for loop counter.       
 *         * x: Current x coordinate
 *         * heightSq: height of rectangle squared
 * 
 * @param numRects numRects we are going to use to estimate the area under the curve. This defines
 * how big our problem size will be. This is the n in KERNEL(n).
 * 
 * @param width of rectangle
 * 
 * @param areas Pre allocated array that will contain  areas. --> This array was allocated with 
 * hipMallocManaged() function which is what leads to UnifiedMemory.
 * 
 * @return fills the areas array
 */

#if CUDA_ENABLED
__global__ 
#endif
void calculateAreas(const long numRects, const double width, double *dev_areas) 
{
/* If cuda is enabled calculate the threadId which gives us the index in dev_areas */   
#if CUDA_ENABLED
    /* Calculate threadId for 1D grid 1D block*/
    //int threadId = threadIdx.x + (blockIdx.x * blockDim.x);
    /* Calculate threadId for 2D grid 1D block*/
    int threadId = (blockIdx.y*gridDim.x + blockIdx.x)*blockDim.x + threadIdx.x;
    if(threadId >= numRects)
    {
        return;
    }
#elif OPENMP_ENABLED
    #pragma omp parallel for
#endif

#if !CUDA_ENABLED
    /* Define the for loop if cuda is not enable. This is used in both the serial and openmp version */
    for(int threadId = 0;threadId < numRects;threadId++)
#endif
    {
        double x = threadId * width;
        double heightSq = 1 - (x*x);
        double height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt(heightSq));
        dev_areas[threadId] = (width * height);   

        /* Add Extra computations in order to be able to see the performance difference between CPU and GPU */
        x = sqrt((float)threadId) * pow(width,3);
        heightSq = 1 - (x*x);
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt((float)heightSq));
        dev_areas[threadId] = (width * height); 

        x = threadId * pow(width,3);
        heightSq = 1 - (x*x);
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt((float)heightSq));
        dev_areas[threadId] = (width * height); 

        x = threadId * pow(width,3);
        heightSq = 1 - (sqrt((float)x)*pow(width,3));
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt((float)heightSq));
        dev_areas[threadId] = (width * height); 

        x = sqrt((float)x) * sqrt((float)x);
        heightSq = 1 - (pow(x,4)*x);
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt((float)heightSq));
        dev_areas[threadId] = (width * height); 

        x = threadId * width;
        heightSq = 1 - (x*x);
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt((float)heightSq));
        dev_areas[threadId] = (width * pow(x,0.5)); 

        x = threadId * width;
        heightSq = 1 - (x*x);
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt(heightSq));
        dev_areas[threadId] = (width * height); 

        x = sqrt((float)threadId) * pow(width,3);
        heightSq = 1 - (x*x);
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt((float)heightSq));
        dev_areas[threadId] = (width * height); 

        x = threadId * pow(width,3);
        heightSq = 1 - (x*x);
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt((float)heightSq));
        dev_areas[threadId] = (width * height); 

        x = threadId * pow(width,3);
        heightSq = 1 - (sqrt((float)x)*pow(width,3));
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt((float)heightSq));
        dev_areas[threadId] = (width * height); 

        x = sqrt((float)x) * sqrt((float)x);
        heightSq = 1 - (pow(x,4)*x);
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt((float)heightSq));
        dev_areas[threadId] = (width * height); 

        x = threadId * width;
        heightSq = 1 - (x*x);
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt((float)heightSq));
        dev_areas[threadId] = (width * pow(x,0.5)); 

        x = threadId * width;
        heightSq = 1 - (x*x);
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt(heightSq));
        dev_areas[threadId] = (width * height); 

        x = sqrt((float)threadId) * pow(width,3);
        heightSq = 1 - (x*x);
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt((float)heightSq));
        dev_areas[threadId] = (width * height); 

        x = threadId * pow(width,3);
        heightSq = 1 - (x*x);
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt((float)heightSq));
        dev_areas[threadId] = (width * height); 

        x = threadId * pow(width,3);
        heightSq = 1 - (sqrt((float)x)*pow(width,3));
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt((float)heightSq));
        dev_areas[threadId] = (width * height); 

        x = sqrt((float)x) * sqrt((float)x);
        heightSq = 1 - (pow(x,4)*x);
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt((float)heightSq));
        dev_areas[threadId] = (width * height); 

        x = threadId * width;
        heightSq = 1 - (x*x);
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt((float)heightSq));
        dev_areas[threadId] = (width * pow(x,0.5)); 

        x = threadId * width;
        heightSq = 1 - (x*x);
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt(heightSq));
        dev_areas[threadId] = (width * height); 

        x = sqrt((float)threadId) * pow(width,3);
        heightSq = 1 - (x*x);
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt((float)heightSq));
        dev_areas[threadId] = (width * height); 

        x = threadId * pow(width,3);
        heightSq = 1 - (x*x);
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt((float)heightSq));
        dev_areas[threadId] = (width * height); 

        x = threadId * pow(width,3);
        heightSq = 1 - (sqrt((float)x)*pow(width,3));
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt((float)heightSq));
        dev_areas[threadId] = (width * height); 

        x = sqrt((float)x) * sqrt((float)x);
        heightSq = 1 - (pow(x,4)*x);
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt((float)heightSq));
        dev_areas[threadId] = (width * height); 

        x = threadId * width;
        heightSq = 1 - (x*x);
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt((float)heightSq));
        dev_areas[threadId] = (width * pow(x,0.5)); 

        x = threadId * width;
        heightSq = 1 - (x*x);
        height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt(heightSq));
        dev_areas[threadId] = (width * height); 
    }
}

void calculateArea(const long numRects, double *area) {

    double *hostAreas;
    double *deviceAreas;
    double *unifiedAreas;

    float milliseconds;
    int i;

/////////////////////////////// MEMORY ALLOCATION SECTION ////////////////////////////////////////

/* If CUDA is enabled allocate memory in device either using hipMalloc or hipMallocManaged */
#if CUDA_ENABLED
    hipError_t err;

    if(getGridDim(numRects) >= 65535)
    {
        fprintf(stderr, "Error: WAY TOO MANY RECTANGLES. Do you really want to compute more than 4.3979123e+12 rectangles!!!! Please input less rectangles");
        return;
    }
    std::cout << "Grid Dimensions = " << getGridDim(numRects) << std::endl;

    #if UNIFIEDMEM_ENABLED
        printf("Unified Memory is Enabled. Allocating using hipMallocManaged \n");
        err = hipMallocManaged(&unifiedAreas, numRects * sizeof(double));
    #else
        printf("Unified Memory is NOT Enabled. Allocating using hipMalloc \n");
        err = hipMalloc(&deviceAreas, numRects * sizeof(double));
    #endif

    /* Check for error in device memory allocation */
    if (err != hipSuccess) 
    {
        fprintf(stderr, "hipMalloc or hipMallocManaged failed: %s\n", hipGetErrorString(err));
    }

/* If CUDA is not enabled we are running on the CPU either serially or with openmp so we allocate memory in the host */
#else 
    hostAreas = (double*)malloc(numRects * sizeof(double));
    if (hostAreas == NULL) 
    {
        fprintf(stderr, "malloc failed!\n");
    }
#endif


/////////////////////////////// KERNEL CALL SECTION ////////////////////////////////////////

/* If CUDA is enabled do the kernel and reduce call either with unifiedMemory or with device memory*/
#if CUDA_ENABLED
    /* Start all cudaEvents so we can record timings */
    hipEvent_t startKernel, startReduce, stopKernel, stopReduce, stopAll;
    hipEventCreate(&startKernel);
    hipEventCreate(&startReduce);

    hipEventCreate(&stopKernel);
    hipEventCreate(&stopReduce);
    hipEventCreate(&stopAll);

    hipEventRecord(startKernel);
    #if UNIFIEDMEM_ENABLED
        calculateAreas KERNEL(numRects) (numRects, (1.0 / numRects), unifiedAreas);
    #else
        calculateAreas KERNEL(numRects) (numRects, (1.0 / numRects), deviceAreas);
    #endif
    hipEventRecord(stopKernel);

    hipEventRecord(startReduce);
    #if UNIFIEDMEM_ENABLED
        (*area) = thrust::reduce(thrust::cuda::par, unifiedAreas, unifiedAreas + numRects);
    #else
        (*area) = thrust::reduce(thrust::cuda::par, deviceAreas, deviceAreas + numRects);
    #endif
    hipEventRecord(stopReduce);
    hipEventRecord(stopAll);

    hipEventSynchronize(stopKernel);
    hipEventSynchronize(stopReduce);
    hipEventSynchronize(stopAll);

    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, startKernel, stopKernel);
    std::cout << "Kernel execution time = " << milliseconds/1000 << "seconds" << std::endl;    
    hipEventElapsedTime(&milliseconds, startReduce, stopReduce);
    std::cout << "Thrust reduce time = " << milliseconds/1000 << "seconds" << std::endl;
    hipEventElapsedTime(&milliseconds, startKernel, stopAll);
    std::cout << "Total kernel + reduce time = " << milliseconds/1000 << "seconds" << std::endl; 

    hipFree(deviceAreas);
    hipFree(unifiedAreas);
/* If CUDA is not enabled calculateAreas is not a kernel but a normal function. */
#else 
    calculateAreas KERNEL(numRects) (numRects, (1.0 / numRects), hostAreas);
    (*area) = 0.0;
    for (i = 0; i < numRects; i++) 
    {
        (*area) += hostAreas[i];
    }

    free(hostAreas);
#endif

///////////////////// GPU OR CPU FREE THE MEMORY ////////////////////

}

#if CUDA_ENABLED
void printDeviceInfo()
{
    int device;
    struct hipDeviceProp_t props;

    hipGetDevice(&device);
    hipGetDeviceProperties(&props, device);

    std::cout << "Device info: " <<std::endl;
    std::cout << "Name: " << props.name <<std::endl;
    std::cout << "version: " << props.major << "," <<  props.minor <<std::endl;
}
#endif