
#include <hip/hip_runtime.h>
#include <stdio.h> /* fprintf() */
#include <float.h> /* DBL_EPSILON and LDBL_DIG */
#include <math.h> /* sqrt() */

__global__ void calculateAreas(const int myNumRects, const double width,
    const int myDispl, double *dev_areas) {
  const int blockId = (blockIdx.x) +
    (blockIdx.y * gridDim.x) +
    (blockIdx.z * gridDim.x * gridDim.y);
  const int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) +
    (threadIdx.x) +
    (threadIdx.y * blockDim.x) +
    (threadIdx.z * (blockDim.x * blockDim.y));
  const double x = ((myDispl + threadId) * width);
  const double heightSq = (1.0 - (x * x));
  /* Prevent nan value for sqrt() */
  const double height = (heightSq < DBL_EPSILON) ? (0.0) : (sqrt(heightSq));

  if (threadId < (myDispl + myNumRects)) {
    dev_areas[threadId] = (width * height);
  }
}

void calculateArea(const int myNumRects, const double width, const int myDispl,
    double *area) {
  const int gridDimX = 1;
  const int gridDimY = 1;
  const int gridDimZ = 1;
  const int blockDimX = 1;
  const int blockDimY = 1;
  const int blockDimZ = myNumRects;
  const dim3 dimGrid(gridDimX, gridDimY, gridDimZ);
  const dim3 dimBlock(blockDimX, blockDimY, blockDimZ);
  double *areas = (double*)malloc(myNumRects * sizeof(double));
  double *dev_areas;
  hipError_t err;
  int i = 0;

  if (areas == NULL) {
    fprintf(stderr, "malloc failed!\n");
  }

  err = hipMalloc((void**)&dev_areas, (myNumRects * sizeof(double)));

  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
  }

  calculateAreas<<<dimGrid, dimBlock>>>(myNumRects, width, myDispl, dev_areas);

  err = hipMemcpy(areas, dev_areas, (myNumRects * sizeof(double)),
    hipMemcpyDeviceToHost);

  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
  }

  (*area) = 0.0;
  for (i = 0; i < myNumRects; i++) {
    (*area) += areas[i];
  }

  hipFree(dev_areas);

  free(areas);
}
